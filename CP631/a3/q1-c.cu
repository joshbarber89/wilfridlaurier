
#include "hip/hip_runtime.h" /* CUDA runtime API */
#include "cstdio"
#include "math.h"

#define N 4096

// For Error Checking, print out erorrs
void error_check(hipError_t cudaStatus, const char* file, int line)
{
    if (cudaStatus != hipSuccess) {
        ::fprintf(stderr, "CUDA ERROR at %s[%d] : %s\n", file, line, hipGetErrorString(cudaStatus));
        abort();
    }
}
#define CUDA_CHECK(err) do { error_check(err, __FILE__, __LINE__); } while(0)

// Atomic Operation to find MIN
__device__ __forceinline__ void atomicMinFloat(float* addr, float value) {
    atomicMin((int*)addr, __float_as_int(value));
}

__global__ void saxpy_gpu(float *x1i, float *x2i, float *y1i, float *y2i) {
    // Global var for all threads
    __shared__ float minV;

    //grab thread id
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    if (i< (N * N)) {
        //initialize values
        float dist, x1, x2, y1, y2;

        float dist_x, dist_y;
        // Get all points
        x1 = x1i[i];
        y1 = y1i[i];
        x2 = x2i[i];
        y2 = y2i[i];

        dist = 2.0f;
        minV = 2.0f;
        // Find distance
        if (x1 != x2 && y1 != y2) {
            // dist = sqrt(pow((x2 - x1),2) + pow((y2 - y1),2));
            dist_x = x2 - x1;
            dist_y = y2 - y1;
            dist = (dist_x * dist_x) + (dist_y * dist_y);
        }
        // Wait for all operations to complete
        __syncthreads();
        // Assign min value
        atomicMinFloat(&minV, dist);
        __syncthreads();
        // assign global min value to output
        if (i == 0)
            x1i[0] = minV;

    }
}
float saxpy_cpu(float *pointX, float *pointY) {
    int i, j;
    float min_dist = 2.0f;
    float x1, y1, x2, y2, dist;

    float dist_x, dist_y;

    for (i = 0; i < N; i++) {
        x1 = pointX[i];
        y1 = pointY[i];

        for (j = 0; j < N; j++) {
            x2 = pointX[j];
            y2 = pointY[j];
            if (x2 != x1 && y2 != y1) {
                //dist = sqrt(pow((x2 - x1),2) + pow((y2 - y1),2));
                dist_x = x2 - x1;
                dist_y = y2 - y1;
                dist = (dist_x * dist_x) + (dist_y * dist_y);
                if (dist < min_dist) {
                    min_dist = dist;
                }
            }
        }
    }
    return min_dist;
}

int main(int argc, char *argv[]) {
    /* arrays for computation on host and device*/
    float *x, *y;
    float *all_combinations_host_x1;
    float *all_combinations_host_x2;
    float *all_combinations_host_y1;
    float *all_combinations_host_y2;
    float *all_combinations_device_x1;
    float *all_combinations_device_x2;
    float *all_combinations_device_y1;
    float *all_combinations_device_y2;

    int i, j, blockSize, nBlocks;


    /* allocate arrays on host */
    x = (float *)malloc(N * sizeof(float));
    y = (float *)malloc(N * sizeof(float));

    all_combinations_host_x1 = (float *)malloc(N * N * sizeof(float));
    all_combinations_host_x2 = (float *)malloc(N * N * sizeof(float));
    all_combinations_device_x1 = (float *)malloc(N * N * sizeof(float));
    all_combinations_device_x2 = (float *)malloc(N * N * sizeof(float));

    all_combinations_host_y1 = (float *)malloc(N * N * sizeof(float));
    all_combinations_host_y2 = (float *)malloc(N * N * sizeof(float));
    all_combinations_device_y1 = (float *)malloc(N * N * sizeof(float));
    all_combinations_device_y2 = (float *)malloc(N * N * sizeof(float));

    /* allocate arrays on device */
    CUDA_CHECK(hipMalloc((void **) &all_combinations_device_x1, N * N * sizeof(float)));
    CUDA_CHECK(hipMalloc((void **) &all_combinations_device_x2, N * N * sizeof(float)));
    CUDA_CHECK(hipMalloc((void **) &all_combinations_device_y1, N * N * sizeof(float)));
    CUDA_CHECK(hipMalloc((void **) &all_combinations_device_y2, N * N * sizeof(float)));

    // Assign particles
    for ( i = 0; i < N; i++) {
        x[i] = rand() / (float)RAND_MAX;
        y[i] = rand() / (float)RAND_MAX;
    }

    // All combinations of x and y within four arrays
    float x1,y1, x2, y2;
    int count = 0;
    for ( i = 0; i < N; i++) {
        x1 = x[i];
        y1 = y[i];
        for ( j = 0; j < N; j++) {
            x2 = x[j];
            y2 = y[j];

            all_combinations_host_x1[count] = x1;
            all_combinations_host_y1[count] = y1;
            all_combinations_host_x2[count] = x2;
            all_combinations_host_y2[count] = y2;
            count++;
        }

    }
    /* copy arrays to device memory */
    CUDA_CHECK(hipMemcpy(all_combinations_device_x1, all_combinations_host_x1, N * N * sizeof(float), hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(all_combinations_device_x2, all_combinations_host_x2, N * N * sizeof(float), hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(all_combinations_device_y1, all_combinations_host_y1, N * N * sizeof(float), hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(all_combinations_device_y2, all_combinations_host_y2, N * N * sizeof(float), hipMemcpyHostToDevice));

    /* set up device execution configuration */
    blockSize = 512;
    nBlocks = (N * N) / blockSize + ((N*N) % blockSize > 0);

    /* Time kernal execution */
    hipEvent_t start, stop;
    float kernel_timer;

    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start, 0);
    /* Call kernal */
    saxpy_gpu<<<nBlocks, blockSize>>>(all_combinations_device_x1, all_combinations_device_x2, all_combinations_device_y1, all_combinations_device_y2);
    /* Check for Errors */
    CUDA_CHECK(hipPeekAtLastError());
    CUDA_CHECK(hipDeviceSynchronize());

    /* Complete time inteval */
    hipEventRecord(stop, 0);
    hipEventSynchronize( stop );
    hipEventElapsedTime( &kernel_timer, start, stop );

    /* spit out Kernal time results */
    printf("Kernel took %f ms\n",kernel_timer);
    hipEventDestroy(start);
    hipEventDestroy(stop);


    /* retrieve results from device (synchronous) */
    hipMemcpy(all_combinations_host_x1, all_combinations_device_x1, N * N * sizeof(float), hipMemcpyDeviceToHost);

    /* guarantee synchronization */
    hipDeviceSynchronize();

    /* Error check with CPU serial */
    float error_check = saxpy_cpu(x, y);

    /* spit out results */
    printf("Min distance GPU calculated is: %f\n", all_combinations_host_x1[0]);

    printf("Min distance CPU serial calculated is : %f\n", error_check);

    /* free memory */
    hipFree(all_combinations_host_x1);
    hipFree(all_combinations_host_x2);
    hipFree(all_combinations_host_y1);
    hipFree(all_combinations_host_y2);
    hipFree(all_combinations_device_x1);
    hipFree(all_combinations_device_x2);
    hipFree(all_combinations_device_y1);
    hipFree(all_combinations_device_y2);
    free(x);
    free(y);

    return 0;
}
