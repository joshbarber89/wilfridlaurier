
#include "hip/hip_runtime.h" /* CUDA runtime API */
#include "cstdio"
#include "math.h"

#define N 65536

// For Error Checking, print out erorrs
void error_check(hipError_t cudaStatus, const char* file, int line)
{
    if (cudaStatus != hipSuccess) {
        ::fprintf(stderr, "CUDA ERROR at %s[%d] : %s\n", file, line, hipGetErrorString(cudaStatus));
        abort();
    }
}
#define CUDA_CHECK(err) do { error_check(err, __FILE__, __LINE__); } while(0)

// Atomic Operation to find MIN
__device__ __forceinline__ void atomicMinFloat(float* addr, float value) {
    atomicMin((int*)addr, __float_as_int(value));
}


__global__ void saxpy_gpu(float *pointX, float *pointY) {
    // Global var for all threads
    __shared__ float minV;

    //grab thread id
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    if (i<N) {
        //initialize values
        float dist, x1, x2, y1, y2;
        int j;

        float local_min = 2.0f;

        float dist_x, dist_y;

        // set global min
        minV = 2.0f;
        // get point
        x2 = pointX[i];
        y2 = pointY[i];
        // compare point to other points
        for (j = 0; j < N; j++) {
            x1 = pointX[j];
            y1 = pointY[j];

            if (x2 != x1 && y2 != y1) {
                //dist = sqrt(pow((x2 - x1),2) + pow((y2 - y1),2));
                dist_x = x2 - x1;
                dist_y = y2 - y1;
                dist = (dist_x * dist_x) + (dist_y * dist_y);

                if (dist < local_min) {
                    local_min = dist;
                }
            }
        }
        // wait for all other threads to complete
        __syncthreads();
        // assign min value
        atomicMinFloat(&minV, local_min);
        __syncthreads();
        // assign global min val to output
        if (i == 0) {
            pointX[0] = minV;
        }

    }
}

float saxpy_cpu(float *pointX, float *pointY) {
    int i, j;
    float min_dist = 2.0f;
    float x1, y1, x2, y2, dist;

    float dist_x, dist_y;

    for (i = 0; i < N; i++) {
        x1 = pointX[i];
        y1 = pointY[i];

        for (j = 0; j < N; j++) {
            x2 = pointX[j];
            y2 = pointY[j];
            if (x2 != x1 && y2 != y1) {
                // dist = sqrt(pow((x2 - x1),2) + pow((y2 - y1),2));
                dist_x = x2 - x1;
                dist_y = y2 - y1;
                dist = (dist_x * dist_x) + (dist_y * dist_y);
                if (dist < min_dist) {
                    min_dist = dist;
                }
            }
        }
    }
    return min_dist;
}

int main(int argc, char *argv[]) {
    float *x_host, *y_host;   /* arrays for computation on host*/
    float *x_dev, *y_dev;     /* arrays for computation on device */
    float *min_dist; /* array for results */

    size_t memsize;
    int i, blockSize, nBlocks;

    memsize = N * sizeof(float);

    /* allocate arrays on host */

    x_host = (float *)malloc(memsize);
    y_host = (float *)malloc(memsize);
    min_dist = (float *)malloc(memsize);

    /* allocate arrays on device */
    CUDA_CHECK(hipMalloc((void **) &x_dev, memsize));
    CUDA_CHECK(hipMalloc((void **) &y_dev, memsize));


    // Add particle values
    for ( i = 0; i < N; i++) {
        x_host[i] = rand() / (float)RAND_MAX;
        y_host[i] = rand() / (float)RAND_MAX;
    }

    /* copy arrays to device memory */
    CUDA_CHECK(hipMemcpy(x_dev, x_host, memsize, hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(y_dev, y_host, memsize, hipMemcpyHostToDevice));

    /* set up device execution configuration */
    blockSize = 512;
    nBlocks = N / blockSize + (N % blockSize > 0);

    /* Time kernal execution */
    hipEvent_t start, stop;
    float kernel_timer;

    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start, 0);

    /* Call kernal */
    saxpy_gpu<<<nBlocks, blockSize>>>(x_dev, y_dev);
    /* Check for Errors */
    CUDA_CHECK(hipPeekAtLastError());
    CUDA_CHECK(hipDeviceSynchronize());

    /* Complete time inteval */
    hipEventRecord(stop, 0);
    hipEventSynchronize( stop );
    hipEventElapsedTime( &kernel_timer, start, stop );

    /* spit out Kernal time results */
    printf("Kernel took %f ms\n",kernel_timer);
    hipEventDestroy(start);
    hipEventDestroy(stop);

    /* Error check with CPU serial */
    float error_check = saxpy_cpu(x_host, y_host);

    /* retrieve results from device (synchronous) */
    hipMemcpy(min_dist, x_dev, memsize, hipMemcpyDeviceToHost);

    /* guarantee synchronization */
    hipDeviceSynchronize();

    /* spit out results */
    printf("Min distance GPU calculated is: %f\n", min_dist[0]);

    printf("Min distance CPU serial calculated is : %f\n", error_check);

    /* free memory */
    hipFree(x_dev);
    hipFree(y_dev);
    free(x_host);
    free(y_host);
    free(min_dist);

    return 0;
}
