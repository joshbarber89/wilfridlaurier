#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h" /* CUDA runtime API */
#include "cstdio"
#include "math.h"

#define N 65536

float saxpy_cpu(float *pointX, float *pointY) {
    // Create vars
    int i;
    int j;
    float dist, y2, x2;
    float min_dist = 2.0f;
    for (i = 0; i < N; i++) {
        // Grab first point then compare to the rest
        float x1 = pointX[i];
        float y1 = pointY[i];

        for (j = 0; j < N; j++) {
            // Grab second point and compute
            x2 = pointX[j];
            y2 = pointY[j];
            if (x2 != x1 && y2 != y1) {
                dist = sqrt(pow((x2 - x1),2) + pow((y2 - y1),2));
                //float dist_x = x2 - x1;
                //float dist_y = y2 - y1;
                //dist = (dist_x * dist_x) + (dist_y * dist_y);

                //Find min
                if (dist < min_dist) {
                    min_dist = dist;
                }
            }
        }
    }
    return min_dist;
}

int main(int argc, char *argv[]) {
    // Set x and y pointers
    float *x, *y;

    size_t memsize;
    int i;

    // Initialize data mmemory
    memsize = N * sizeof(float);

    x = (float *)malloc(memsize);
    y = (float *)malloc(memsize);

    // Set x y coords
    for ( i = 0; i < N; i++) {
        x[i] = rand() / (float)RAND_MAX;
        y[i] = rand() / (float)RAND_MAX;
    }

    // begin clock
    clock_t begin = clock();

    // Call algorithm
    float min_dist = saxpy_cpu(x, y);

    // Ending stats and freeing memory
    clock_t end = clock();
    double time_spent = (double)(end - begin) / CLOCKS_PER_SEC;

    printf("Min distance: %f\n", min_dist);
    printf("Time of execution: %fs", time_spent);

    free(x);
    free(y);

    return 0;
}
